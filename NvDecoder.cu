#include <iostream>
#include <algorithm>
#include <chrono>
#include <npp.h>
#include "NvDecoder.hpp"



#define __D(fmt, args...) printf("" fmt, ## args)
#define __I(fmt, args...) printf("" fmt, ## args)
#define __E(fmt, args...) fprintf(stderr, "" fmt, ## args)

#define _stricmp strcasecmp
#define _stat64 stat64

#ifndef ck
#define ck(call) check((call), #call, __FILE__, __LINE__)
template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
    if (result) {
        fprintf(stderr, "Decoder error at %s:%d code=%d \"%s\" \n", file, line,
                static_cast<unsigned int>(result), func);
        exit(EXIT_FAILURE);
    }
}
#endif


#define START_TIMER auto start = std::chrono::high_resolution_clock::now();
#define STOP_TIMER(print_message) std::cout << print_message << \
    std::chrono::duration_cast<std::chrono::milliseconds>( \
    std::chrono::high_resolution_clock::now() - start).count() \
    << " ms " << std::endl;

#define CUDA_DRVAPI_CALL( call )                                                                                                 \
    do                                                                                                                           \
    {                                                                                                                            \
        hipError_t err__ = call;                                                                                                   \
        if (err__ != hipSuccess)                                                                                               \
        {                                                                                                                        \
            const char *szErrName = NULL;                                                                                        \
            hipDrvGetErrorName(err__, &szErrName);                                                                                   \
            std::ostringstream errorLog;                                                                                         \
            errorLog << "CUDA driver API error " << szErrName ;                                                                  \
            throw NVDECException::makeNVDECException(errorLog.str(), err__, __FUNCTION__, __FILE__, __LINE__);                   \
        }                                                                                                                        \
    }                                                                                                                            \
    while (0)





/* Return value from HandleVideoSequence() are interpreted as   :
*  0: fail, 1: succeeded, > 1: override dpb size of parser (set by CUVIDPARSERPARAMS::ulMaxNumDecodeSurfaces while creating parser)
*/
int NvDecoder::handleNvSequence(CUVIDEOFORMAT *pVideoFormat)
{
    START_TIMER
    m_videoInfo.str("");
    m_videoInfo.clear();
    m_videoInfo << "Video Input Information" << std::endl
        << "\tCodec        : " << getCodecString(pVideoFormat->codec) << std::endl
        << "\tFrame rate   : " << pVideoFormat->frame_rate.numerator << "/" << pVideoFormat->frame_rate.denominator
            << " = " << 1.0 * pVideoFormat->frame_rate.numerator / pVideoFormat->frame_rate.denominator << " fps" << std::endl
        << "\tSequence     : " << (pVideoFormat->progressive_sequence ? "Progressive" : "Interlaced") << std::endl
        << "\tCoded size   : [" << pVideoFormat->coded_width << ", " << pVideoFormat->coded_height << "]" << std::endl
        << "\tDisplay area : [" << pVideoFormat->display_area.left << ", " << pVideoFormat->display_area.top << ", "
            << pVideoFormat->display_area.right << ", " << pVideoFormat->display_area.bottom << "]" << std::endl
        << "\tChroma       : " << getChromaString(pVideoFormat->chroma_format) << std::endl
        << "\tBit depth    : " << pVideoFormat->bit_depth_luma_minus8 + 8
    ;
    m_videoInfo << std::endl;

    nDecodeSurface = pVideoFormat->min_num_decode_surfaces;

    CUVIDDECODECAPS decodecaps = {
        .eCodecType      = pVideoFormat->codec,
        .eChromaFormat   = pVideoFormat->chroma_format,
        .nBitDepthMinus8 = pVideoFormat->bit_depth_luma_minus8
    };

    CUDA_DRVAPI_CALL(hipCtxPushCurrent(m_cuContext));
    NVDEC_API_CALL(cuvidGetDecoderCaps(&decodecaps));
    CUDA_DRVAPI_CALL(hipCtxPopCurrent(NULL));

    if(!decodecaps.bIsSupported) {
        NVDEC_THROW_ERROR("Codec not supported on this GPU", hipErrorNotSupported);
        return nDecodeSurface;
    }

    if ((pVideoFormat->coded_width > decodecaps.nMaxWidth) ||
        (pVideoFormat->coded_height > decodecaps.nMaxHeight)){

        std::ostringstream errorString;
        errorString << std::endl
                    << "Resolution          : " << pVideoFormat->coded_width << "x" << pVideoFormat->coded_height << std::endl
                    << "Max Supported (wxh) : " << decodecaps.nMaxWidth << "x" << decodecaps.nMaxHeight << std::endl
                    << "Resolution not supported on this GPU";

        const std::string cErr = errorString.str();
        NVDEC_THROW_ERROR(cErr, hipErrorNotSupported);
        return nDecodeSurface;
    }

    if ((pVideoFormat->coded_width>>4)*(pVideoFormat->coded_height>>4) > decodecaps.nMaxMBCount){

        std::ostringstream errorString;
        errorString << std::endl
                    << "MBCount             : " << (pVideoFormat->coded_width >> 4)*(pVideoFormat->coded_height >> 4) << std::endl
                    << "Max Supported mbcnt : " << decodecaps.nMaxMBCount << std::endl
                    << "MBCount not supported on this GPU";

        const std::string cErr = errorString.str();
        NVDEC_THROW_ERROR(cErr, hipErrorNotSupported);
        return nDecodeSurface;
    }

    if (m_nWidth && m_nLumaHeight && m_nChromaHeight) {

        // cuvidCreateDecoder() has been called before, and now there's possible config change
        return nvReconfigureDecoder(pVideoFormat);
    }

    nvCreateDecoder(pVideoFormat);

    STOP_TIMER("Session Initialization Time: ");
    return nDecodeSurface;
}

int NvDecoder::nvCreateDecoder(CUVIDEOFORMAT *pVideoFormat)
{
    // eCodec has been set in the constructor (for parser). Here it's set again for potential correction
    m_eCodec          = pVideoFormat->codec;
    m_eChromaFormat   = pVideoFormat->chroma_format;
    m_nBitDepthMinus8 = pVideoFormat->bit_depth_luma_minus8;
    m_nBPP            = m_nBitDepthMinus8 > 0 ? 2 : 1;

    if (m_eChromaFormat == cudaVideoChromaFormat_420) {
        m_eOutputFormat = pVideoFormat->bit_depth_luma_minus8 ? cudaVideoSurfaceFormat_P016 : cudaVideoSurfaceFormat_NV12;
    } else if (m_eChromaFormat == cudaVideoChromaFormat_444) {
        m_eOutputFormat = pVideoFormat->bit_depth_luma_minus8 ? cudaVideoSurfaceFormat_YUV444_16Bit : cudaVideoSurfaceFormat_YUV444;
    }
    m_videoFormat = *pVideoFormat;

    CUVIDDECODECREATEINFO nvI = { 0 };
    nvI.CodecType      = pVideoFormat->codec;
    nvI.ChromaFormat   = pVideoFormat->chroma_format;
    nvI.OutputFormat   = m_eOutputFormat;
    nvI.bitDepthMinus8 = pVideoFormat->bit_depth_luma_minus8;
    if (pVideoFormat->progressive_sequence)
        nvI.DeinterlaceMode = cudaVideoDeinterlaceMode_Weave;
    else
        nvI.DeinterlaceMode = cudaVideoDeinterlaceMode_Adaptive;
    nvI.ulNumOutputSurfaces = 2;
    // With PreferCUVID, JPEG is still decoded by CUDA while video is decoded by NVDEC hardware
    nvI.ulCreationFlags     = cudaVideoCreate_PreferCUVID;
    nvI.ulNumDecodeSurfaces = nDecodeSurface;
    nvI.vidLock  = m_ctxLock;
    nvI.ulWidth  = pVideoFormat->coded_width;
    nvI.ulHeight = pVideoFormat->coded_height;
    if (m_nMaxWidth < (int)pVideoFormat->coded_width)
        m_nMaxWidth = pVideoFormat->coded_width;
    if (m_nMaxHeight < (int)pVideoFormat->coded_height)
        m_nMaxHeight = pVideoFormat->coded_height;
    nvI.ulMaxWidth  = m_nMaxWidth;
    nvI.ulMaxHeight = m_nMaxHeight;

    if (!(m_cropRect.r && m_cropRect.b) &&
        !(m_resizeDim.w && m_resizeDim.h))
    {
        m_nWidth      = pVideoFormat->display_area.right - pVideoFormat->display_area.left;
        m_nLumaHeight = pVideoFormat->display_area.bottom - pVideoFormat->display_area.top;
        nvI.ulTargetWidth  = pVideoFormat->coded_width;
        nvI.ulTargetHeight = pVideoFormat->coded_height;
    }
    else
    {
        if (m_resizeDim.w && m_resizeDim.h) {
            nvI.display_area.left   = pVideoFormat->display_area.left;
            nvI.display_area.top    = pVideoFormat->display_area.top;
            nvI.display_area.right  = pVideoFormat->display_area.right;
            nvI.display_area.bottom = pVideoFormat->display_area.bottom;
            m_nWidth      = m_resizeDim.w;
            m_nLumaHeight = m_resizeDim.h;
        }

        __I("%d,%d,%d,%d. %dx%d\n",
            m_cropRect.l,
            m_cropRect.t,
            m_cropRect.r,
            m_cropRect.b,
            m_nWidth, m_nLumaHeight);

        if (m_cropRect.r && m_cropRect.b) {
            nvI.display_area.left    = m_cropRect.l;
            nvI.display_area.top     = m_cropRect.t;
            nvI.display_area.right   = m_cropRect.r;
            nvI.display_area.bottom  = m_cropRect.b;
            m_nWidth      = m_cropRect.r - m_cropRect.l;
            m_nLumaHeight = m_cropRect.b - m_cropRect.t;
        }
        nvI.ulTargetWidth  = m_nWidth;
        nvI.ulTargetHeight = m_nLumaHeight;
    }

    m_nSurfaceHeight = nvI.ulTargetHeight;
    m_nSurfaceWidth = nvI.ulTargetWidth;

    m_chromaHeight_factor = getChromaHeightFactor(nvI.ChromaFormat);
    m_nChromaHeight = (int)(m_nLumaHeight * m_chromaHeight_factor);
    m_nNumChromaPlanes = getChromaPlaneCount(nvI.ChromaFormat);
    //m_nSurfaceChromaHeight = (int)(m_nSurfaceHeight * m_chromaHeight_factor);

    m_displayRect.b = nvI.display_area.bottom;
    m_displayRect.t = nvI.display_area.top;
    m_displayRect.l = nvI.display_area.left;
    m_displayRect.r = nvI.display_area.right;


    m_videoInfo << "Video Decoding Params:" << std::endl
        << "\tNum Surfaces : " << nvI.ulNumDecodeSurfaces << std::endl
        << "\tCrop         : [" << nvI.display_area.left << ", " << nvI.display_area.top << ", "
        << nvI.display_area.right << ", " << nvI.display_area.bottom << "]" << std::endl
        << "\tResize       : " << nvI.ulTargetWidth << "x" << nvI.ulTargetHeight << std::endl
        << "\tDeinterlace  : " << std::vector<const char *>{"Weave", "Bob", "Adaptive"}[nvI.DeinterlaceMode]
    ;
    m_videoInfo << std::endl;

    CUDA_DRVAPI_CALL(hipCtxPushCurrent(m_cuContext));
    NVDEC_API_CALL(cuvidCreateDecoder(&m_hDecoder, &nvI));
    CUDA_DRVAPI_CALL(hipCtxPopCurrent(NULL));
    return 0;
}

int NvDecoder::nvReconfigureDecoder(CUVIDEOFORMAT *pVideoFormat)
{
    if (pVideoFormat->bit_depth_luma_minus8   != m_videoFormat.bit_depth_luma_minus8 ||
        pVideoFormat->bit_depth_chroma_minus8 != m_videoFormat.bit_depth_chroma_minus8)
    {
        NVDEC_THROW_ERROR("Reconfigure Not supported for bit depth change", hipErrorNotSupported);
    }

    if (pVideoFormat->chroma_format != m_videoFormat.chroma_format) {
        NVDEC_THROW_ERROR("Reconfigure Not supported for chroma format change", hipErrorNotSupported);
    }

    bool bDecodeResChange   = !(pVideoFormat->coded_width == m_videoFormat.coded_width &&
                                pVideoFormat->coded_height == m_videoFormat.coded_height);

    bool bDisplayRectChange = !(pVideoFormat->display_area.bottom == m_videoFormat.display_area.bottom &&
                                pVideoFormat->display_area.top    == m_videoFormat.display_area.top &&
                                pVideoFormat->display_area.left   == m_videoFormat.display_area.left &&
                                pVideoFormat->display_area.right  == m_videoFormat.display_area.right);

    nDecodeSurface = pVideoFormat->min_num_decode_surfaces;

    if ((pVideoFormat->coded_width > m_nMaxWidth) ||
        (pVideoFormat->coded_height > m_nMaxHeight))
    {
        // For VP9, let driver  handle the change if new width/height > maxwidth/maxheight
        if ((m_eCodec != cudaVideoCodec_VP9) || m_bReconfigExternal) {
            NVDEC_THROW_ERROR("Reconfigure Not supported when width/height > maxwidth/maxheight", hipErrorNotSupported);
        }
        return 1;
    }

    if (!bDecodeResChange && !m_bReconfigExtPPChange) {
        // if the coded_width/coded_height hasn't changed but display resolution has changed, then need to update width/height for
        // correct output without cropping. Example : 1920x1080 vs 1920x1088
        if (bDisplayRectChange)
        {
            m_nWidth           = pVideoFormat->display_area.right - pVideoFormat->display_area.left;
            m_nLumaHeight      = pVideoFormat->display_area.bottom - pVideoFormat->display_area.top;
            m_nChromaHeight    = int(m_nLumaHeight * getChromaHeightFactor(pVideoFormat->chroma_format));
            m_nNumChromaPlanes = getChromaPlaneCount(pVideoFormat->chroma_format);
        }

        // no need for reconfigureDecoder(). Just return
        return 1;
    }

    CUVIDRECONFIGUREDECODERINFO nvC = { 0 };

    nvC.ulWidth = m_videoFormat.coded_width = pVideoFormat->coded_width;
    nvC.ulHeight = m_videoFormat.coded_height = pVideoFormat->coded_height;

    // Dont change display rect and get scaled output from decoder. This will help display app to present apps smoothly
    nvC.display_area.bottom = m_displayRect.b;
    nvC.display_area.top    = m_displayRect.t;
    nvC.display_area.left   = m_displayRect.l;
    nvC.display_area.right  = m_displayRect.r;
    nvC.ulTargetWidth  = m_nSurfaceWidth;
    nvC.ulTargetHeight = m_nSurfaceHeight;

    // If external reconfigure is called along with resolution change even if post processing params is not changed,
    // do full reconfigure params update
    if ((m_bReconfigExternal && bDecodeResChange) ||
        m_bReconfigExtPPChange)
    {
        // update display rect and target resolution if requested explicitely
        m_bReconfigExternal    = false;
        m_bReconfigExtPPChange = false;
        m_videoFormat = *pVideoFormat;

        if (!(m_cropRect.r && m_cropRect.b) &&
            !(m_resizeDim.w && m_resizeDim.h))
        {
            m_nWidth      = pVideoFormat->display_area.right - pVideoFormat->display_area.left;
            m_nLumaHeight = pVideoFormat->display_area.bottom - pVideoFormat->display_area.top;
            nvC.ulTargetWidth  = pVideoFormat->coded_width;
            nvC.ulTargetHeight = pVideoFormat->coded_height;
        }
        else
        {
            if (m_resizeDim.w && m_resizeDim.h) {
                nvC.display_area.left   = pVideoFormat->display_area.left;
                nvC.display_area.top    = pVideoFormat->display_area.top;
                nvC.display_area.right  = pVideoFormat->display_area.right;
                nvC.display_area.bottom = pVideoFormat->display_area.bottom;
                m_nWidth      = m_resizeDim.w;
                m_nLumaHeight = m_resizeDim.h;
            }

            if (m_cropRect.r && m_cropRect.b) {
                nvC.display_area.left    = m_cropRect.l;
                nvC.display_area.top     = m_cropRect.t;
                nvC.display_area.right   = m_cropRect.r;
                nvC.display_area.bottom  = m_cropRect.b;
                m_nWidth      = m_cropRect.r - m_cropRect.l;
                m_nLumaHeight = m_cropRect.b - m_cropRect.t;
            }
            nvC.ulTargetWidth  = m_nWidth;
            nvC.ulTargetHeight = m_nLumaHeight;
        }

        m_nChromaHeight = int(m_nLumaHeight * getChromaHeightFactor(pVideoFormat->chroma_format));
        m_nNumChromaPlanes = getChromaPlaneCount(pVideoFormat->chroma_format);
        m_nSurfaceHeight = nvC.ulTargetHeight;
        m_nSurfaceWidth = nvC.ulTargetWidth;
        m_displayRect.b = nvC.display_area.bottom;
        m_displayRect.t = nvC.display_area.top;
        m_displayRect.l = nvC.display_area.left;
        m_displayRect.r = nvC.display_area.right;
    }

    nvC.ulNumDecodeSurfaces = nDecodeSurface;

    START_TIMER
    CUDA_DRVAPI_CALL(hipCtxPushCurrent(m_cuContext));
    NVDEC_API_CALL(cuvidReconfigureDecoder(m_hDecoder, &nvC));
    CUDA_DRVAPI_CALL(hipCtxPopCurrent(NULL));
    STOP_TIMER("Session Reconfigure Time: ");

    return nDecodeSurface;
}

int NvDecoder::setReconfigParams(const Rect *pCropRect, const Dim *pResizeDim)
{
    m_bReconfigExternal    = true;
    m_bReconfigExtPPChange = false;

    if (pCropRect)
    {
        if (!((pCropRect->t == m_cropRect.t)  &&
              (pCropRect->l == m_cropRect.l)  &&
              (pCropRect->b == m_cropRect.b) &&
              (pCropRect->r == m_cropRect.r))) {
            m_bReconfigExtPPChange = true;
            m_cropRect = *pCropRect;
        }
    }

    if (pResizeDim)
    {
        if (!((pResizeDim->w == m_resizeDim.w) &&
              (pResizeDim->h == m_resizeDim.h))) {
            m_bReconfigExtPPChange = true;
            m_resizeDim = *pResizeDim;
        }
    }

    // Clear existing output buffers of different size
    uint8_t *pFrame = NULL;
    while (!m_vpFrame.empty())
    {
        pFrame = m_vpFrame.back();
        m_vpFrame.pop_back();
        if (m_bUseDeviceFrame)
        {
            // d_frame
            CUDA_DRVAPI_CALL(hipCtxPushCurrent(m_cuContext));
            CUDA_DRVAPI_CALL(hipFree((hipDeviceptr_t)pFrame));
            CUDA_DRVAPI_CALL(hipCtxPopCurrent(NULL));
        }
        else
        {
            // h_frame
            delete pFrame;
        }
    }
    m_vpFrameRet.clear();

    return 0;
}

/* Return value from HandlePictureDecode() are interpreted as:
*  0: fail, >=1: succeeded
*/
int NvDecoder::handleNvDecode(CUVIDPICPARAMS *pPicParams) {
    if (!m_hDecoder)
    {
        NVDEC_THROW_ERROR("Decoder not initialized.", hipErrorNotInitialized);
        return 0;
    }
    m_nPicNumInDecodeOrder[pPicParams->CurrPicIdx] = m_nDecodePicCnt++;
    NVDEC_API_CALL(cuvidDecodePicture(m_hDecoder, pPicParams));
    return 1;
}

/* Return value from HandlePictureDisplay() are interpreted as:
*  0: fail, >=1: succeeded
*/
int NvDecoder::handleNvPostProc(CUVIDPARSERDISPINFO *pDispInfo) {
    CUVIDPROCPARAMS nvPr = {};
    nvPr.progressive_frame = pDispInfo->progressive_frame;
    nvPr.second_field      = pDispInfo->repeat_first_field + 1;
    nvPr.top_field_first   = pDispInfo->top_field_first;
    nvPr.unpaired_field    = pDispInfo->repeat_first_field < 0;
    nvPr.output_stream     = m_cuvidStream;

    hipDeviceptr_t  d_srcFrame = 0;
    unsigned int d_srcPitch = 0;

    NVDEC_API_CALL(cuvidMapVideoFrame(m_hDecoder,
                                      pDispInfo->picture_index,
                                      &d_srcFrame,
                                      &d_srcPitch,
                                      &nvPr));

    CUVIDGETDECODESTATUS nvS = {};
    hipError_t result = cuvidGetDecodeStatus(m_hDecoder,
                                           pDispInfo->picture_index,
                                           &nvS);
    if (result == hipSuccess &&
        (nvS.decodeStatus == cuvidDecodeStatus_Error ||
         nvS.decodeStatus == cuvidDecodeStatus_Error_Concealed))
    {
        printf("Decode Error occurred for picture %d\n", m_nPicNumInDecodeOrder[pDispInfo->picture_index]);
    }

    uint8_t *pDecodedFrame = nullptr;
    int frameSize; // = getFrameSize(); // NV12
    frameSize = d_srcPitch*m_nLumaHeight*3; // RGBI
    /* if (oformat == IMAGE_RGBI) { */
    /*     frameSize = d_srcPitch*m_nLumaHeight*3; */
    /* } */

    {

        std::lock_guard<std::mutex> lock(m_mtxVPFrame);
        if ((unsigned)++m_nDecodedFrame > m_vpFrame.size())
        {
            // Not enough frames in stock
            m_nFrameAlloc++;
            uint8_t *pFrame = NULL;
            if (m_bUseDeviceFrame)
            {
                // GPU DEVICE memory if m_bUseDeviceFrame:1
                CUDA_DRVAPI_CALL(hipCtxPushCurrent(m_cuContext));
                if (m_bDeviceFramePitched)
                {
                    CUDA_DRVAPI_CALL(hipMemAllocPitch((hipDeviceptr_t *)&pFrame,
                                                     &m_nDeviceFramePitch,
                                                     d_srcPitch*3, //m_nWidth * m_nBPP,
                                                     m_nLumaHeight*3, //m_nLumaHeight + (m_nChromaHeight * m_nNumChromaPlanes),
                                                     16));
                }
                else
                {
                    CUDA_DRVAPI_CALL(hipMalloc((hipDeviceptr_t *)&pFrame, frameSize));
                }
                CUDA_DRVAPI_CALL(hipCtxPopCurrent(NULL));
            }
            else
            {
                // CPU HOST memory if m_bUseDeviceFrame:0
                pFrame = new uint8_t[frameSize];
            }
            m_vpFrame.push_back(pFrame);
        }
        pDecodedFrame = m_vpFrame[m_nDecodedFrame - 1];
    }

    CUDA_DRVAPI_CALL(hipCtxPushCurrent(m_cuContext));
    // TODO start

    if (oformat == IMAGE_NV12) {
        hip_Memcpy2D m = { 0 };
        m.srcMemoryType = hipMemoryTypeDevice;
        m.srcDevice     = d_srcFrame;
        m.srcPitch      = d_srcPitch;
        m.Height        = m_nLumaHeight;

        m.dstDevice     = (hipDeviceptr_t)(m.dstHost = pDecodedFrame);
        m.dstMemoryType = m_bUseDeviceFrame ? hipMemoryTypeDevice : hipMemoryTypeHost;
        m.dstPitch      = m_nDeviceFramePitch ? m_nDeviceFramePitch : m_nWidth * m_nBPP;
        m.WidthInBytes  = m_nWidth * m_nBPP;
        CUDA_DRVAPI_CALL(hipMemcpyParam2DAsync(&m, m_cuvidStream));

        m.srcDevice = (hipDeviceptr_t)((uint8_t *)d_srcFrame + m.srcPitch * m_nSurfaceHeight);
        m.dstDevice = (hipDeviceptr_t)(m.dstHost = pDecodedFrame + m.dstPitch * m_nLumaHeight);
        m.Height = m_nChromaHeight; // TODO: (size_t)(m_nSurfaceHeight * m_chromaHeight_factor)
        CUDA_DRVAPI_CALL(hipMemcpyParam2DAsync(&m, m_cuvidStream));

        if (m_nNumChromaPlanes == 2)
        {
            m.srcDevice = (hipDeviceptr_t)((uint8_t *)d_srcFrame + m.srcPitch * m_nSurfaceHeight * 2); // TODO
            m.dstDevice = (hipDeviceptr_t)(m.dstHost = pDecodedFrame + m.dstPitch * m_nLumaHeight * 2); // TODO
            m.Height = m_nChromaHeight; // TODO
            CUDA_DRVAPI_CALL(hipMemcpyParam2DAsync(&m, m_cuvidStream));
        }
        CUDA_DRVAPI_CALL(hipStreamSynchronize(m_cuvidStream));
    } else {
        // 2RGBi
        if (m_d_RGBi_frame == NULL) {
            hipMalloc(&m_d_RGBi_frame, d_srcPitch*m_nLumaHeight*3*sizeof(float));
        }


        NppiSize d_size = {(int)d_srcPitch, (int)m_nLumaHeight};
        Npp8u* y_plane = (Npp8u*)d_srcFrame;
        Npp8u* uv_plane = y_plane + d_srcPitch * m_nLumaHeight;
        Npp8u* src_planes[2] = {y_plane,uv_plane};
        nppiNV12ToRGB_8u_P2C3R(src_planes,
                               d_srcPitch,
                               (Npp8u*)m_d_RGBi_frame,
                               d_srcPitch*3,
                               d_size);

        if (oformat == IMAGE_RGBI)
        {
            hip_Memcpy2D m = { 0 };
            m.srcMemoryType = hipMemoryTypeDevice;
            m.srcDevice     = (hipDeviceptr_t)m_d_RGBi_frame;
            m.srcPitch      = d_srcPitch*3;
            m.Height        = m_nLumaHeight;

            m.dstDevice     = (hipDeviceptr_t)(m.dstHost = pDecodedFrame);
            m.dstMemoryType = m_bUseDeviceFrame ? hipMemoryTypeDevice : hipMemoryTypeHost;
            //m.dstPitch      = m_nDeviceFramePitch ? m_nDeviceFramePitch : m_nWidth * m_nBPP; //TODO: m_nDeviceFramePitch should be rgbi
            m.dstPitch      = m_nDeviceFramePitch ? m_nDeviceFramePitch : m_nWidth * 3;
            m.WidthInBytes  = m_nWidth * 3;
            CUDA_DRVAPI_CALL(hipMemcpyParam2DAsync(&m, m_cuvidStream));
            CUDA_DRVAPI_CALL(hipStreamSynchronize(m_cuvidStream));
        }
        else if (oformat == IMAGE_RGB)
        {
            // 2RGBp
            if (m_d_RGBp_frame == NULL) {
                hipMalloc(&m_d_RGBp_frame, d_srcPitch*m_nLumaHeight*3*sizeof(float));
            }

            Npp8u* r_plane = (Npp8u*)m_d_RGBp_frame;
            Npp8u* g_plane = r_plane + d_srcPitch*m_nLumaHeight;
            Npp8u* b_plane = r_plane + d_srcPitch*m_nLumaHeight*2;
            Npp8u* rgb_planes[3] = {r_plane,g_plane,b_plane};
            nppiCopy_8u_C3P3R((Npp8u*)m_d_RGBi_frame,
                              d_srcPitch*3,
                              rgb_planes,
                              d_srcPitch,
                              d_size);

            hip_Memcpy2D m = { 0 };
            m.srcMemoryType = hipMemoryTypeDevice;
            m.srcDevice     = (hipDeviceptr_t)m_d_RGBp_frame;
            m.srcPitch      = d_srcPitch;
            m.Height        = m_nLumaHeight*3;

            m.dstDevice     = (hipDeviceptr_t)(m.dstHost = pDecodedFrame);
            m.dstMemoryType = m_bUseDeviceFrame ? hipMemoryTypeDevice : hipMemoryTypeHost;
            m.dstPitch      = m_nDeviceFramePitch ? m_nDeviceFramePitch : m_nWidth;
            m.WidthInBytes  = m_nWidth;
            CUDA_DRVAPI_CALL(hipMemcpyParam2DAsync(&m, m_cuvidStream));
            CUDA_DRVAPI_CALL(hipStreamSynchronize(m_cuvidStream));
        }



    }


    // TODO end
    CUDA_DRVAPI_CALL(hipCtxPopCurrent(NULL));

    if ((int)m_vTimestamp.size() < m_nDecodedFrame) {
        m_vTimestamp.resize(m_vpFrame.size());
    }
    m_vTimestamp[m_nDecodedFrame - 1] = pDispInfo->timestamp;

    NVDEC_API_CALL(cuvidUnmapVideoFrame(m_hDecoder, d_srcFrame));
    return 1;
}

NvDecoder::NvDecoder(uint16_t instanceId, Rect *pCropRect, Dim  *pResizeDim, hipCtx_t cuContext)
{

    iGpu = (int)instanceId;

    ck(hipInit(0));

    ck(hipGetDeviceCount(&nGpu));
    if (iGpu < 0 || iGpu >= nGpu) {
        __E("GPU Decoder instance out of range. Should be within[0, %d] \n",
            nGpu-1);
        return;
    }

    ck(hipDeviceGet(&cuDevice, iGpu));
    ck(hipDeviceGetName(m_deviceName, sizeof(m_deviceName), cuDevice));
    __I("GPU Decoder instance:%d in use: %s \n",
        iGpu, m_deviceName);

    if (cuContext == NULL) {
        ck(hipCtxCreate(&m_selfCtx, 0, cuDevice));
        cuContext = m_selfCtx;
    }
    nvCreateParser(cuContext,
                   false, // m_bUseDeviceFrame
                   cudaVideoCodec_H264,
                   NULL, false, false,
                   pCropRect, pResizeDim);
}

void NvDecoder::nvCreateParser(hipCtx_t cuContext,
                               bool bUseDeviceFrame,
                               cudaVideoCodec eCodec,
                               std::mutex *pMutex,
                               bool bLowLatency,
                               bool bDeviceFramePitched,
                               const Rect *pCropRect,
                               const Dim *pResizeDim,
                               int maxWidth,
                               int maxHeight)
{
    m_cuContext = cuContext;
    m_bUseDeviceFrame = bUseDeviceFrame;
    m_eCodec = eCodec;
    m_pMutex = pMutex;
    m_bDeviceFramePitched = bDeviceFramePitched;
    m_nMaxWidth = maxWidth;
    m_nMaxHeight = maxHeight;

    if (pCropRect) m_cropRect = *pCropRect;
    if (pResizeDim) m_resizeDim = *pResizeDim;

    NVDEC_API_CALL(cuvidCtxLockCreate(&m_ctxLock, cuContext));

    CUVIDPARSERPARAMS nvPa = {};
    nvPa.CodecType              = eCodec;
    nvPa.ulMaxNumDecodeSurfaces = 1;
    nvPa.ulMaxDisplayDelay      = bLowLatency ? 0 : 1;
    nvPa.pUserData           = this;
    nvPa.pfnSequenceCallback = handleNvSequenceIsr;
    nvPa.pfnDecodePicture    = handleNvDecodeIsr;
    nvPa.pfnDisplayPicture   = handleNvPostProcIsr;

    if (m_pMutex) m_pMutex->lock();
    NVDEC_API_CALL(cuvidCreateVideoParser(&m_hParser, &nvPa));
    if (m_pMutex) m_pMutex->unlock();
}

NvDecoder::~NvDecoder() {

    START_TIMER
    hipCtxPushCurrent(m_cuContext);
    hipCtxPopCurrent(NULL);

    if (m_hParser) {
        cuvidDestroyVideoParser(m_hParser);
    }

    if (m_hDecoder) {
        if (m_pMutex) m_pMutex->lock();
        cuvidDestroyDecoder(m_hDecoder);
        if (m_pMutex) m_pMutex->unlock();
    }

    std::lock_guard<std::mutex> lock(m_mtxVPFrame);
    if (m_vpFrame.size() != m_nFrameAlloc)
    {
        //LOG(WARNING) << "nFrameAlloc(" << m_nFrameAlloc << ") != m_vpFrame.size()(" << m_vpFrame.size() << ")";
    }
    for (uint8_t *pFrame : m_vpFrame)
    {
        if (m_bUseDeviceFrame)
        {
            if (m_pMutex) m_pMutex->lock();
            hipCtxPushCurrent(m_cuContext);
            hipFree((hipDeviceptr_t)pFrame);
            hipCtxPopCurrent(NULL);
            if (m_pMutex) m_pMutex->unlock();
        }
        else
        {
            delete[] pFrame;
        }
    }
    cuvidCtxLockDestroy(m_ctxLock);
    STOP_TIMER("Session Deinitialization Time: ");

    if (m_selfCtx) {
        hipCtxDestroy(m_selfCtx);
        m_selfCtx = NULL;
    }

}

int NvDecoder::decode(const uint8_t *bitstream, int bitstreamBytes,
                       uint8_t ***pppFrame, int *pnFrameReturned,
                       uint32_t flags, int64_t **ppTimestamp, int64_t timestamp, hipStream_t stream)
{
    if (!m_hParser)
    {
        NVDEC_THROW_ERROR("Parser not initialized.", hipErrorNotInitialized);
        return -1;
    }

    CUVIDSOURCEDATAPACKET packet = {0};
    packet.payload      = bitstream;
    packet.payload_size = bitstreamBytes;
    packet.flags        = flags | CUVID_PKT_TIMESTAMP;
    packet.timestamp    = timestamp;
    if (!bitstream || bitstreamBytes == 0) {
        packet.flags |= CUVID_PKT_ENDOFSTREAM;
    }

    m_nDecodedFrame = 0;
    m_cuvidStream = stream;
    if (m_pMutex) m_pMutex->lock();
    NVDEC_API_CALL(cuvidParseVideoData(m_hParser, &packet));
    if (m_pMutex) m_pMutex->unlock();
    m_cuvidStream = 0;

    if (m_nDecodedFrame > 0)
    {
        if (pppFrame)
        {
            m_vpFrameRet.clear();
            std::lock_guard<std::mutex> lock(m_mtxVPFrame);
            m_vpFrameRet.insert(m_vpFrameRet.begin(),
                                m_vpFrame.begin(),
                                m_vpFrame.begin() + m_nDecodedFrame);
            *pppFrame = &m_vpFrameRet[0];
        }
        if (ppTimestamp)
        {
            *ppTimestamp = &m_vTimestamp[0];
        }
    }
    if (pnFrameReturned)
    {
        *pnFrameReturned = m_nDecodedFrame;
    }
    return 0;
}

int NvDecoder::decode_lockFrame(const uint8_t *bitstream, int bitstreamBytes, uint8_t ***pppFrame, int *pnFrameReturned, uint32_t flags, int64_t **ppTimestamp, int64_t timestamp, hipStream_t stream)
{
    auto ret = decode(bitstream, bitstreamBytes, pppFrame, pnFrameReturned, flags, ppTimestamp, timestamp, stream);
    if (ret) {
        return ret;
    }

    std::lock_guard<std::mutex> lock(m_mtxVPFrame);
    m_vpFrame.erase(m_vpFrame.begin(), m_vpFrame.begin() + m_nDecodedFrame);
    return ret;
}

void NvDecoder::unlockFrame(uint8_t **ppFrame, int nFrame)
{
    std::lock_guard<std::mutex> lock(m_mtxVPFrame);
    m_vpFrame.insert(m_vpFrame.end(), &ppFrame[0], &ppFrame[nFrame]);
}

#if 0
extern "C"
void *
TestH264DecoderCreate(
    uint16_t instanceId,
    TestRect *lCropRect,
    TestDim  *lResizeDim)
{
    Rect _rect, *pCropRect = NULL;
    Dim  _dim,  *pResizeDim = NULL;

    if (lCropRect) {
        pCropRect = &_rect;
        pCropRect->l = lCropRect->left;
        pCropRect->t = lCropRect->top;
        pCropRect->r = lCropRect->right;
        pCropRect->b = lCropRect->bottom;
    }

    if (lResizeDim) {
        pResizeDim = &_dim;
        pResizeDim->w = lResizeDim->w;
        pResizeDim->h = lResizeDim->h;
    }

    auto decoder = new NvDecoder(instanceId, pCropRect, pResizeDim);
    return static_cast<void*>(decoder);
}

extern "C"
void
TestH264DecoderDestroy(void *decoder_handle)
{
    auto decoder = static_cast<NvDecoder*>(decoder_handle);
    delete(decoder);
}

extern "C"
TestMediaStatus
TestH264Decode(
    void *decoder_handle,
    const TestBitstreams *bitstreams,
    TestImage *output_image)
{
    auto decoder = static_cast<NvDecoder*>(decoder_handle);

    if (output_image->colorStd) {
        auto oformat = static_cast<NvDecoder::ImageFormat_t>(output_image->colorStd);
        if (decoder->oformat != oformat) {
            __D("convert to format: %d:%s \n", oformat,
                (oformat == NvDecoder::IMAGE_NV12)  ? "nv12" :
                (oformat == NvDecoder::IMAGE_YUV)  ? "yuv" :
                (oformat == NvDecoder::IMAGE_Y)    ? "y" :
                (oformat == NvDecoder::IMAGE_RGB)  ? "rgb" :
                (oformat == NvDecoder::IMAGE_BGR)  ? "bgr" :
                (oformat == NvDecoder::IMAGE_RGBI) ? "rgbi" :
                (oformat == NvDecoder::IMAGE_BGRI) ? "bgri" :
                "unknown"
                );
            decoder->oformat = oformat;
        }
    }

    uint8_t **ppFrame;
    int   nFrameReturned = 0;
    auto ret = decoder->decode(bitstreams->bitstream, bitstreams->bitstreamBytes,
                               &ppFrame, &nFrameReturned);
    if (nFrameReturned == 0) {
        return -1;
    }

    for (int i = 0; i < nFrameReturned; i++) {
        output_image->img          = ppFrame[i];
        output_image->bytePerPixel = decoder->getBPP();
        output_image->width        = decoder->getWidth();
        output_image->height       = decoder->getHeight();
        output_image->colorStd = static_cast<TestImageFormat_t>(decoder->oformat);
    }

    return 0;
}

extern "C"
void
printH264Info(void *decoder_handle)
{
    auto decoder = static_cast<NvDecoder*>(decoder_handle);

    std::vector <std::string> aszDecodeOutFormat = { "NV12", "P016", "YUV444", "YUV444P16" };
    /* if (bOutPlanar) { */
    /*     aszDecodeOutFormat[0] = "iyuv"; */
    /*     aszDecodeOutFormat[1] = "yuv420p16"; */
    /* } */

    __I("%s \nDecodeOutFormat: %s \n",
        decoder->getVideoInfo().c_str(),
        aszDecodeOutFormat[decoder->getOutputFormat()].c_str()
        );
}




extern "C"
void
printDecoderCapability()
{
    ck(hipInit(0));
    int nGpu = 0;
    ck(hipGetDeviceCount(&nGpu));
    std::cout << "Decoder Capability" << std::endl << std::endl;
    const char *aszCodecName[] = {"JPEG", "MPEG1", "MPEG2", "MPEG4", "H264", "HEVC", "HEVC", "HEVC", "HEVC", "HEVC", "HEVC", "VC1", "VP8", "VP9", "VP9", "VP9"};
    const char *aszChromaFormat[] = { "4:0:0", "4:2:0", "4:2:2", "4:4:4" };
    //char strOutputFormats[64];
    cudaVideoCodec aeCodec[] = { cudaVideoCodec_JPEG, cudaVideoCodec_MPEG1, cudaVideoCodec_MPEG2, cudaVideoCodec_MPEG4, cudaVideoCodec_H264, cudaVideoCodec_HEVC,
        cudaVideoCodec_HEVC, cudaVideoCodec_HEVC, cudaVideoCodec_HEVC, cudaVideoCodec_HEVC, cudaVideoCodec_HEVC, cudaVideoCodec_VC1, cudaVideoCodec_VP8,
        cudaVideoCodec_VP9, cudaVideoCodec_VP9, cudaVideoCodec_VP9 };
    int anBitDepthMinus8[] = {0, 0, 0, 0, 0, 0, 2, 4, 0, 2, 4, 0, 0, 0, 2, 4};

    cudaVideoChromaFormat aeChromaFormat[] = { cudaVideoChromaFormat_420, cudaVideoChromaFormat_420, cudaVideoChromaFormat_420, cudaVideoChromaFormat_420,
        cudaVideoChromaFormat_420, cudaVideoChromaFormat_420, cudaVideoChromaFormat_420, cudaVideoChromaFormat_420, cudaVideoChromaFormat_444, cudaVideoChromaFormat_444,
        cudaVideoChromaFormat_444, cudaVideoChromaFormat_420, cudaVideoChromaFormat_420, cudaVideoChromaFormat_420, cudaVideoChromaFormat_420, cudaVideoChromaFormat_420 };

    for (int iGpu = 0; iGpu < nGpu; iGpu++) {

        hipCtx_t cuContext = NULL;
        //createCudaContext(&cuContext, iGpu, 0);
        hipDevice_t cuDevice = 0;
        ck(hipDeviceGet(&cuDevice, iGpu));
        char szDeviceName[80];
        ck(hipDeviceGetName(szDeviceName, sizeof(szDeviceName), cuDevice));
        std::cout << "GPU in use: " << szDeviceName << std::endl;
        ck(hipCtxCreate(&cuContext, 9, cuDevice));


        for (int i = 0; i < sizeof(aeCodec) / sizeof(aeCodec[0]); i++) {

            CUVIDDECODECAPS decodeCaps = {};
            decodeCaps.eCodecType = aeCodec[i];
            decodeCaps.eChromaFormat = aeChromaFormat[i];
            decodeCaps.nBitDepthMinus8 = anBitDepthMinus8[i];

            cuvidGetDecoderCaps(&decodeCaps);

            std::string outputFormat = "";
            if (decodeCaps.nOutputFormatMask & (1U << cudaVideoSurfaceFormat_NV12)) {
                outputFormat = "NV12";
            }

            if (decodeCaps.nOutputFormatMask & (1U << cudaVideoSurfaceFormat_P016)) {
                outputFormat = "P016";
            }

            if (decodeCaps.nOutputFormatMask & (1U << cudaVideoSurfaceFormat_YUV444)) {
                outputFormat = "YUV444";
            }

            if (decodeCaps.nOutputFormatMask & (1U << cudaVideoSurfaceFormat_YUV444_16Bit)) {
                outputFormat = "YUV444P16";
            }


            if (decodeCaps.bIsSupported) {
                __I("Codec: %-5s  BitDepth: %-3u MaxWidth: %-6u MaxHeight: %-6u MaxMBCount: %-9u Format: %-s %s\n",
                    aszCodecName[i],
                    decodeCaps.nBitDepthMinus8 + 8,
                    decodeCaps.nMaxWidth, decodeCaps.nMaxHeight,
                    decodeCaps.nMaxMBCount,
                    aszChromaFormat[decodeCaps.eChromaFormat], &outputFormat[0]);
            }
        }

        ck(hipCtxDestroy(cuContext));
    }
}
#endif
