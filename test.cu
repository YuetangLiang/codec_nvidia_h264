#include <iostream>
#include <algorithm>
#include <thread>
#include <hip/hip_runtime.h>
#include <sstream>
#include <iostream>
#include <fstream>
#include <iomanip>

#include "test_demuxer.h"
#include "test_viddec.h"
//#include "test_nvjpg.h"

#define __D(fmt, args...) printf("" fmt, ## args)
#define __I(fmt, args...) printf("" fmt, ## args)
#define __E(fmt, args...) fprintf(stderr, "" fmt, ## args)

#define _stricmp strcasecmp
#define _stat64 stat64

#ifndef ck
#define ck(call) check((call), #call, __FILE__, __LINE__)
template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
    if (result) {
        fprintf(stderr, "Decoder error at %s:%d code=%d \"%s\" \n", file, line,
            static_cast<unsigned int>(result), func);
        exit(EXIT_FAILURE);
    }
}
#endif




int main(int argc, char **argv)
{
    int nFrame = 0;
    bool bDecodeOutSemiPlanar = false;
    void *encoder = NULL;
    bool bOutPlanar = false;
    TestRect lCropRect = {
        .left   = 30,
        .top    = 10,
        .right  = 640,
        .bottom = 480
    };
    TestDim  lResizeDim = {
        .w = 640,
        .h = 480
    };

    //printDecoderCapability();

    auto szInFilePath = argc > 1 ? std::string(argv[1]) : "input.mp4";
    auto demuxer = TestDemuxerCreate(&szInFilePath[0]);
    auto decoder = TestH264DecoderCreate(0, &lCropRect, &lResizeDim);
    TestBitstreams bitstreams;

    do {
        TestImage om;
        //om.colorStd = TEST_IMAGE_NV12;
        //om.colorStd = TEST_IMAGE_RGBI;
        om.colorStd = TEST_IMAGE_RGB;

        TestDemux(demuxer,
                   &bitstreams.bitstream,
                   &bitstreams.bitstreamBytes);

        auto ret = TestH264Decode(decoder, &bitstreams, &om);
        if (ret != TEST_MEDIA_STATUS_OK) {
            // bitstream insufficient
            continue;
        }

        if (!nFrame) {
            printH264Info(decoder);
        }

        //bDecodeOutSemiPlanar = (om.colorStd == TEST_IMAGE_YUV);
        if (om.img) {
            if (om.colorStd == TEST_IMAGE_NV12) {
                // dump NV12
                char nv12_path[100];
                sprintf(nv12_path, "%d_NV12_%dx%d.yuv",
                        nFrame,
                        om.width, om.height);
                auto nv12File = fopen(nv12_path, "w+");

                if (fwrite(om.img, om.width*om.height*1.5, 1, nv12File) != 1) {
                    return -1;
                }
                fclose(nv12File);

            } else if (om.colorStd == TEST_IMAGE_RGBI) {
                // dump RGBI
                char rgbi_path[100];
                sprintf(rgbi_path, "%d_RGBi_%dx%d.rgb",
                        nFrame,
                        om.width, om.height);
                auto rgbiFile = fopen(rgbi_path, "w+");

                if (fwrite(om.img, om.width*om.height*3, 1, rgbiFile) != 1) {
                    return -1;
                }
                fclose(rgbiFile);
            } else if (om.colorStd == TEST_IMAGE_RGB) {
                // dump RGBp
                char rgbp_path[100];
                sprintf(rgbp_path, "%d_RGBp_%dx%d.rgb",
                        nFrame,
                        om.width, om.height);
                auto rgbpFile = fopen(rgbp_path, "w+");

                if (fwrite(om.img, om.width*om.height*3, 1, rgbpFile) != 1) {
                    return -1;
                }
                fclose(rgbpFile);
            }

            // convert2jpg
            if (!encoder) {
                encoder = TestJpgEncoderCreate(om.colorStd,
                                                om.width, om.height,
                                                0,0);
            }

            TestBitstreams output_bs = {0};
            TestJpgEncode(encoder, &om, &output_bs);

            auto jpg_path = std::to_string(nFrame) + "out.jpg";
            auto outputFile = fopen(&jpg_path[0], "w+");

            if (fwrite(output_bs.bitstream, output_bs.bitstreamBytes, 1, outputFile) != 1) {
                return 0;
            }
            fclose(outputFile);
        }

        if (bOutPlanar && bDecodeOutSemiPlanar) {
            //ConvertSemiplanarToPlanar(ppFrame[i], dec.GetWidth(), dec.GetHeight(), dec.GetBitDepth());
            __D("\n");
        }


        nFrame += 1;

    } while (bitstreams.bitstreamBytes);

    TestDemuxerDestroy(demuxer);

    std::cout << "Total frame decoded: " << nFrame << std::endl;

    return 0;
}
